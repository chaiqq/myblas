#include "hip/hip_runtime.h"
#include "kernels.h"
#include "util.hpp"
#include <iostream>

namespace gpu {
size_t get1DGrid(size_t blockSize, size_t matrixSize) {
    // TODO: complete function
    // given matrixSize, n*n, compute how many blocks do we need
    // assume blocks are arranged in 1D
    // eg: get1DGrid(5, 16) = 4
    // ceiling(matrixSize / blockSize)'
    return (matrixSize + blockSize - 1) / blockSize;
    // -1 是避免 get1DGrid(4, 16)的情况, 多出一个block
    // size_t是个unsigned int
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_matrixMultGlobal(const float *devA, const float *devB, float *devC,
                                        const int size) {
    // 
    // TODO: complete function
    // A thread is in charge of 1 result element in devC
    // threads同时执行这个kernel
    // 目前我们的grid中只有一个block

    // 我是一个thread, 我在block中的相对坐标为(threadIdx.x, threadIdx.y), 我负责计算C矩阵的第row行,第col列
    // 我的绝对坐标需要结合我所在的block的坐标进行计算
    // 计算的过程是一个点乘
    // int row = threadIdx.y;
    // int col = threadIdx.x;
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    // int col = blockIdx.x * blockDim.x + threadIdx.x;
    // NOTE: 注意索引的计算
    // 答案里的row和col怎么看怎么别扭,如果简单地直接映射topology到matrix上, 计算就简单了
    // 修改: 由于这里矩阵使用col-major, fastest-running的坐标是row, 所谓fastest,是相邻内存的内容. 因为要迎合cuBLAS的col major
    // A[i][j]的相邻内存是A[i+1][j], col-major不改变我们习惯的矩阵元素编号. 
    // fastest就是体现在i变动最快, 也就是row号先变动, 一列完成后,再变动col号
    // 在cuda中, thread编号是 (x,y),  (x+1,y)
    //                      (x,y+1),(x+1, y+1)
    // z这个编号应该fastest变的是x号, 所以要与矩阵的fastet对应
    // 当然row = ...y * ..y 那个也可以
    // 
    

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;


    // C(i, k) += A(i, j) * B(j, k), for j = 0 : size, i = row, k = col
    if((row < size) && (col < size)){
        float accumulator = 0.0f;

        for(int j = 0; j < size; ++j){
            accumulator += devA[j * size + row] * devB[col * size + j];
        }

        devC[col * size + row] += accumulator;
    }
}

void executeMatrixMultGlobal(dim3 dimBlock, dim3 dimGrid, float *Ad, float *Bd, float *Cd,
                             const Configuration &config) {
    for (int i = 0; i < config.numRepeats; ++i) {
        kernel_matrixMultGlobal<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
    }
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
template <int TILE_SIZE>
__global__ void kernel_matrixMultTiled(const float *__restrict__ devA,
                                       const float *__restrict__ devB, float *__restrict__ devC,
                                       const size_t size) {
  // TODO: complete function
  __shared__ float shrA[TILE_SIZE][TILE_SIZE];
  __shared__ float shrB[TILE_SIZE][TILE_SIZE];

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int row = blockIdx.y * TILE_SIZE + ty;
  const int col = blockIdx.x * TILE_SIZE + tx; // blockDim.x == TILE_SIZE 这里简化,设block和tile一样大
  

  // NOTE: 详细索引,循环设置的原理见ppt cuda_p3, 第8页, 第13页
  // 答案我感觉i和j的计算是错误的, 目前使用自己觉得舒服的计算
  // C22 = A21*B12 + A22*B22 + A23*B32; 这里的A21是一个tile, 这里有3个phase
  if((row < size) && (col < size)){
      float Celem = 0.0f;
      for(int phase = 0; phase < size / TILE_SIZE; ++phase) {// 假设整个矩阵大小正好整除tile size
        shrA[ty][tx] = devA[ (phase * TILE_SIZE + tx) * size + row ];
        shrB[ty][tx] = devB[ col * size + phase * TILE_SIZE + ty];
        __syncthreads();
        for(int k = 0; k < TILE_SIZE; k++){
            Celem += shrA[ty][k] * shrB[k][tx];
        }
        __syncthreads();
      }
      devC[col * size + row] = Celem;
  }

}
/* Tiled
Elapsed time : 0.00988173 s
operations: 2.68173e+08
Performance: 25.2745 GFlop/s
*/


void executeMatrixMultTiled(dim3 dimBlock, dim3 dimGrid, float *Ad, float *Bd, float *Cd,
                            const Configuration &config) {
    switch (config.tileSize) {
        case 4:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultTiled<4><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 8:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultTiled<8><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 16:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultTiled<16><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 32:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultTiled<32><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
    }
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
template <int TILE_SIZE>
__global__ void kernel_matrixMultCoalesced(const float *__restrict__ devA,
                                           const float *__restrict__ devB, float *__restrict__ devC,
                                           const size_t size) {
  // TODO: complete function
  __shared__ float shrA[TILE_SIZE][TILE_SIZE];
  __shared__ float shrB[TILE_SIZE][TILE_SIZE];

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int row = blockIdx.x * TILE_SIZE + tx;
  const int col = blockIdx.y * TILE_SIZE + ty; // blockDim.x == TILE_SIZE 这里简化,设block和tile一样大
  // 这里的逻辑是, 在GPU中,tx是fastest running index, 要对应A的内存,colmajor的fastest running是沿着列
  // 这样子, 两个neighboring threads, 相差一个tx, 就能access 同样也是相邻的A的两个元素


  // 想要coalesced, 需要在同一个iter中时, 相邻的thread访问连续的一块内存, 而不是同一个thread在不同iter
  // 相邻的thread具有相邻的threadIdx.x, 所以要让x不能stride访问
  if((row < size) && (col < size)){
      float Celem = 0.0f;

      // 可以选择换一种mapping矩阵到block的方式, 想象成将block转置. 去顺应fastest running的那个维度的变化
      for(int phase = 0; phase < size / TILE_SIZE; ++phase) {// 假设整个矩阵大小正好整除tile size
        shrA[ty][tx] = devA[(phase * TILE_SIZE + ty) * size + row]; // access global的时候相邻的thread会合并访存
        shrB[ty][tx] = devB[col * size + phase * TILE_SIZE + tx];
        // 
        // load进来的shrA,shrB是原来的转置
        // 其实也可以shrA[tx][ty] = ... 这样load进来的就是原来形状的A 和B了,相应的celem +=也要改一下
        __syncthreads();
        // shared mem又是row major
        for(int k = 0; k < TILE_SIZE; k++){
            Celem += shrA[k][tx] * shrB[ty][k];
        }
        __syncthreads();
      }
      devC[col * size + row] = Celem;
  }
}
/*
Coalesced:
Elapsed time : 0.00659242 s
operations: 2.68173e+08
Performance: 37.8853 GFlop/s
*/

void executeMatrixMultCoalesced(dim3 dimBlock, dim3 dimGrid, float *Ad, float *Bd, float *Cd,
                                const Configuration &config) {
    switch (config.tileSize) {
        case 4:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultCoalesced<4><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 8:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultCoalesced<8><<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 16:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultCoalesced<16>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 32:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultCoalesced<32>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
    }
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_matrixMultCoalescedDym(const float *__restrict__ devA,
                                              const float *__restrict__ devB,
                                              float *__restrict__ devC, const size_t size) {
  // TODO: complete function
  // shr mem size may be known only at run-time, so compute and pass it as the third parameter when launching the kernel
    const int TILE_SIZE = blockDim.x;
    extern __shared__ float shrA[];
    float *__restrict__ shrB = &shrA[TILE_SIZE * TILE_SIZE]; // shrB从shrA内存的一半处开始

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int row = blockIdx.x * TILE_SIZE + tx;
    const int column = blockIdx.y * TILE_SIZE + ty;
    if ((row < size) && (column < size)) {
        float Celem = 0.0f;

        for (int m = 0; m < size / TILE_SIZE; ++m) {
            // load tiles of A and B to the shared mem.
            shrA[tx + TILE_SIZE * ty] = devA[row + size * (ty + m * TILE_SIZE)]; //shrA[ty][tx]的一维
            shrB[tx + TILE_SIZE * ty] = devB[(tx + m * TILE_SIZE) + column * size];
            __syncthreads();

            for (int j = 0; j < TILE_SIZE; ++j)
                Celem += shrA[tx + TILE_SIZE * j] * shrB[j + TILE_SIZE * ty];
            __syncthreads();
        };

        devC[row + size * column] += Celem;
    }
}


void executeMatrixMultCoalescedDym(dim3 dimBlock, dim3 dimGrid, float *Ad, float *Bd, float *Cd,
                                   const Configuration &config) {
    const size_t shrMemSize = 2 * config.tileSize * config.tileSize * sizeof(float);
    for (int i = 0; i < config.numRepeats; ++i) {
        kernel_matrixMultCoalescedDym<<<dimGrid, dimBlock, shrMemSize>>>(Ad, Bd, Cd,
                                                                         config.matrixSize);
    }
    // NOTE: the 3rd parameter is shrMemSize
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
template <int TILE_SIZE>
__global__ void kernel_matrixMultOverlapped(const float *__restrict__ devA,
                                            const float *__restrict__ devB,
                                            float *__restrict__ devC, const size_t size) {
    // TODO: complete function, prefetch to register
    // load first tile into registers, copy registers to shared memory, barrier
    // load next tile into registers. compute current tile(shr mem), barrier
    // repeat above 2 steps
    // compute last tile
    __shared__ float shrA[TILE_SIZE][TILE_SIZE];
    __shared__ float shrB[TILE_SIZE][TILE_SIZE];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int row = blockIdx.x * TILE_SIZE + tx;
    const int column = blockIdx.y * TILE_SIZE + ty;
    if ((row < size) && (column < size)) {

        float Celem = 0.0f, Aelem = 0.0f, Belem = 0.0f;
        // load the first tile into registers
        Aelem = devA[row + size * ty];
        Belem = devB[tx + column * size];

        for (int m = 0; m < (size / TILE_SIZE) - 1; ++m) {
            // load tiles of A and B to the shared mem.
            shrA[ty][tx] = Aelem;
            shrB[ty][tx] = Belem;
            __syncthreads();

            // load the next tile to the registers
            Aelem = devA[row + size * (ty + (m + 1) * TILE_SIZE)];
            Belem = devB[(tx + (m + 1) * TILE_SIZE) + column * size];

            for (int j = 0; j < TILE_SIZE; ++j)
                Celem += shrA[j][tx] * shrB[ty][j];
            __syncthreads();
        };

        // compute the last tile
        const int m = (size / TILE_SIZE) - 1;
        shrA[ty][tx] = Aelem;
        shrB[ty][tx] = Belem;
        __syncthreads();
        for (int j = 0; j < TILE_SIZE; ++j)
          Celem += shrA[j][tx] * shrB[ty][j];

        devC[row + size * column] += Celem;
    }


}


void executeMatrixMultOverlapped(dim3 dimBlock, dim3 dimGrid, float *Ad, float *Bd, float *Cd,
                                 const Configuration &config) {
    switch (config.tileSize) {
        case 4:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultOverlapped<4>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 8:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultOverlapped<8>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 16:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultOverlapped<16>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
        case 32:
            for (int i = 0; i < config.numRepeats; ++i) {
                kernel_matrixMultOverlapped<32>
                    <<<dimGrid, dimBlock>>>(Ad, Bd, Cd, config.matrixSize);
            }
            break;
    }
    CHECK_ERR;
}

//--------------------------------------------------------------------------------------------------
void matrixMult(float *Ad, float *Bd, float *Cd, const Configuration &config) {
    // TODO: adjust dimBlock and dimGrid
    dim3 dimBlock(config.tileSize, config.tileSize); // tilesize = 8的话, 64 threads
    const size_t Grid1D = get1DGrid(dimBlock.x, config.matrixSize);
    dim3 dimGrid(Grid1D, Grid1D);// assume grid is square

    switch (config.kernelType) {
        case KernelType::KERNEL_GLOBAL:
            executeMatrixMultGlobal(dimBlock, dimGrid, Ad, Bd, Cd, config);
            break;
        case KernelType::KERNEL_TILED:
            executeMatrixMultTiled(dimBlock, dimGrid, Ad, Bd, Cd, config);
            break;
        case KernelType::KERNEL_COALESCED:
            executeMatrixMultCoalesced(dimBlock, dimGrid, Ad, Bd, Cd, config);
            break;
        case KernelType::KERNEL_COALESCED_DYM:
            executeMatrixMultCoalescedDym(dimBlock, dimGrid, Ad, Bd, Cd, config);
            break;
        case KernelType::KERNEL_OVERLAPPED:
            executeMatrixMultOverlapped(dimBlock, dimGrid, Ad, Bd, Cd, config);
            break;
    }
    CHECK_ERR;
}
} // namespace gpu
