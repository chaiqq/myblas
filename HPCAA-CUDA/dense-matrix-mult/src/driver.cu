#include "driver.h"
#include "kernels/kernels.h"
#include "util.hpp"
#include <chrono>
#include <hipblas.h>

float compute(std::vector<float> &C, const std::vector<float> &A, const std::vector<float> &B,
              const Configuration &config) { // 由于hipMemcpy需要传入的src是一个const void *src, 所以传参时, A是一个const

    hipDeviceReset();
    CHECK_ERR; // GPU error occurs but may terminate in another place: to locate the error
    // Check err every time after each GPU API call. save time for debugging
     // errors usually on GPU, CPU does not know
    // if not check, 有可能GPU在执行第三行时,第一行执行完了,但是问题出在第一行
    // 准确定位error的位置

    float *devA{nullptr}, *devB{nullptr}, *devC{nullptr};
    {
      // TODO: Allocate matrices A, B, C on device
      hipMalloc(&devA, A.size() * sizeof(float)); CHECK_ERR;
      hipMalloc(&devB, B.size() * sizeof(float)); CHECK_ERR;
      hipMalloc(&devC, C.size() * sizeof(float)); CHECK_ERR;
    }

    {
      // TODO: Copy the data from host to the device
      // NOTE: You may copy C as well, as it is zeroed, or hipMemset it to zero on the device
      // devA是一个float *的指针. 所以memcpy只要传一个指针即devA即可. 而hipMalloc需要传**, 指向指针的指针, 因此需要传入&devA
      hipMemcpy(devA, A.data(), A.size() * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
      hipMemcpy(devB, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
      hipMemcpy(devC, C.data(), C.size() * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
      // hipMemset(devC, 0, C.size() * sizeof(float));
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float cpuTime{};
    hipEvent_t startTimer{}, stopTimer{};
    hipEventCreate(&startTimer);
    hipEventCreate(&stopTimer);


    // Start computing
    hipEventRecord(startTimer, 0);
    switch (config.kernelType) {
        case KernelType::KERNEL_CPU: {
            auto begin = std::chrono::high_resolution_clock::now();
            // NOTE: repeat loop is inside of cpu::matrixMult
            cpu::matrixMult(C, A, B, config);
            auto end = std::chrono::high_resolution_clock::now();
            cpuTime =
                std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(end - begin).count();
            break;
        }
        case KernelType::KERNEL_CUBLAS: {
          // 调用cublas库计算
            float alpha = 1.0f, beta = 1.0f;
            for (int i = 0; i < config.numRepeats; ++i) {
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, config.matrixSize, config.matrixSize,
                            config.matrixSize, &alpha, devA, config.matrixSize, devB,
                            config.matrixSize, &beta, devC, config.matrixSize);
                            // HIPBLAS_OP_N: non transpose operation
                // NOTE: float是单精度,所以用sgemm
            }
            CHECK_ERR;
            break;
        }
        default: {
            // NOTE: repeat loop is inside of gpu::matrixMult
            gpu::matrixMult(devA, devB, devC, config); // use the self-written mat-mul
            break;
        }
    }

    hipEventRecord(stopTimer, 0);
    hipEventSynchronize(stopTimer);
    CHECK_ERR;
    // NOTE: hipEventSynchronize(stopTimer) is implicit hipDeviceSynchronize() in this context
    // The time measurement does not count memory movements, only focus on kernel performance
    float gpuTime{};
    hipEventElapsedTime(&gpuTime, startTimer, stopTimer);

    // release resources
    hipblasDestroy(handle);
    hipEventDestroy(startTimer);
    hipEventDestroy(stopTimer);

    {
      // TODO: transfer matrix C back, from device to the host
      // NOTE: const_cast<float *>(); generally, sizeof(float) returns 4, i.e. single precision
      hipMemcpy(const_cast<float *> (C.data()), devC, C.size() * sizeof(float), hipMemcpyDeviceToHost);
      CHECK_ERR;
      // NOTE: should be const_cast, otherwise breaks
    }

    {
      // TODO: clean gpu memory
      hipFree(devA); CHECK_ERR;
      hipFree(devB); CHECK_ERR;
      hipFree(devC); CHECK_ERR;
    }
    return (config.kernelType == KernelType::KERNEL_CPU) ? cpuTime : gpuTime;
}
