#include "hip/hip_runtime.h"
#include "kernels.h"
#include <iostream>
#include <map>
#include <sstream>

#define WARP_SIZE 32

namespace err {
std::string PrevFile{};
int PrevLine{0};


void checkErr(const std::string &file, int line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::stringstream stream;
        stream << '\n'
               << file << ", line " << line << ": " << hipGetErrorString(Error) << " (" << Error
               << ")\n";
        if (PrevLine > 0) {
            stream << "Previous CUDA call:" << '\n' << PrevFile << ", line " << PrevLine << '\n';
        }
        throw std::runtime_error(stream.str());
    }
    PrevFile = file;
    PrevLine = line;
#endif
}

void checkCublasStatus(hipblasStatus_t status, const std::string &file, int line) {
    static std::map<hipblasStatus_t, std::string> cublasErrorMap{
        {HIPBLAS_STATUS_SUCCESS, "HIPBLAS_STATUS_SUCCESS"},
        {HIPBLAS_STATUS_NOT_INITIALIZED, "HIPBLAS_STATUS_NOT_INITIALIZED"},
        {HIPBLAS_STATUS_ALLOC_FAILED, "HIPBLAS_STATUS_ALLOC_FAILED"},
        {HIPBLAS_STATUS_INVALID_VALUE, "HIPBLAS_STATUS_INVALID_VALUE"},
        {HIPBLAS_STATUS_ARCH_MISMATCH, "HIPBLAS_STATUS_ARCH_MISMATCH"},
        {HIPBLAS_STATUS_MAPPING_ERROR, "HIPBLAS_STATUS_MAPPING_ERROR"},
        {HIPBLAS_STATUS_EXECUTION_FAILED, "HIPBLAS_STATUS_EXECUTION_FAILED"},
        {HIPBLAS_STATUS_INTERNAL_ERROR, "HIPBLAS_STATUS_INTERNAL_ERROR"}};

    if (status == HIPBLAS_STATUS_SUCCESS) {
        return;
    } else {
        std::stringstream stream;
        stream << file << ", line " << line << ": ";
        if (cublasErrorMap.find(status) != cublasErrorMap.end()) {
            stream << "cublas returned with error: " << cublasErrorMap[status];
        } else {
            stream << "cublas returned with unknown error";
        }
        throw std::runtime_error(stream.str());
    }
}

void checkCusparseStatus(hipsparseStatus_t status, const std::string &file, int line) {
    static std::map<hipsparseStatus_t, std::string> cusparseErrorMap{
        {HIPSPARSE_STATUS_SUCCESS, "HIPSPARSE_STATUS_SUCCESS"},
        {HIPSPARSE_STATUS_NOT_INITIALIZED, "HIPSPARSE_STATUS_NOT_INITIALIZED"},
        {HIPSPARSE_STATUS_ALLOC_FAILED, "HIPSPARSE_STATUS_ALLOC_FAILED"},
        {HIPSPARSE_STATUS_INVALID_VALUE, "HIPSPARSE_STATUS_INVALID_VALUE"},
        {HIPSPARSE_STATUS_ARCH_MISMATCH, "HIPSPARSE_STATUS_ARCH_MISMATCH"},
        {HIPSPARSE_STATUS_MAPPING_ERROR, "HIPSPARSE_STATUS_MAPPING_ERROR"},
        {HIPSPARSE_STATUS_EXECUTION_FAILED, "HIPSPARSE_STATUS_EXECUTION_FAILED"},
        {HIPSPARSE_STATUS_INTERNAL_ERROR, "HIPSPARSE_STATUS_INTERNAL_ERROR"},
        {HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED, "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED"}};

    if (status == HIPSPARSE_STATUS_SUCCESS) {
        return;
    } else {
        std::stringstream stream;
        stream << file << ", line " << line << ": ";
        if (cusparseErrorMap.find(status) != cusparseErrorMap.end()) {
            stream << "cusparse returned with error: " << cusparseErrorMap[status];
        } else {
            stream << "cusparse returned with unknown error";
        }
        throw std::runtime_error(stream.str());
    }
}
} // namespace err


std::string getDeviceName() {
    int deviceId{-1};
    hipGetDevice(&deviceId);

    hipDeviceProp_t devProp{};
    hipGetDeviceProperties(&devProp, deviceId);
    std::stringstream stream;

    stream << devProp.name << ", Compute Capability: " << devProp.major << '.' << devProp.minor;
    return stream.str();
}

size_t get1DGrid(size_t blockSize, size_t size) {
    return (size + blockSize - 1) / blockSize;
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_csrMatVecMult(float *y, const DevCsrMatrix matrix, const float *x) {
  // TODO: T3.2a implement mat-vec multiplication
  // 每个thread负责一行的点积
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < matrix.numRows){
    float dot = 0.0f;
    for(int i = matrix.start[row]; i < matrix.start[row+1]; i++){
        dot += matrix.values[i] * x[matrix.indices[i]];
    }
    y[row] = dot;
  }
}


template <int TILE_SIZE>
__global__ void kernel_csrMatVecMult_vectorized(float *y, const DevCsrMatrix matrix, const float *x) {
  // TODO: H3.1 implement mat-vec multiplication
  // each warp does a row * vector multiplication for one row of matrix
  // 1. assign one warp to a matrix row
  // 2. allocate a shared arry vals[] for the partial results of a block
  // 3. compute one row*vec product in a loop. This time, parallelize the loop over all 32 threads in the warp
  // take care that access to the arrays j and a is coalesced
  // 4. use reduction (binary fan-in) to add up the partial sum in vals[], and add the output to the result vector y

  __shared__ float values[TILE_SIZE];

    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / WARP_SIZE;
    int lane = threadId & (WARP_SIZE - 1); // 相当于mod
    int row = warpId;

    if (row < matrix.numRows) {
        int rowStart = matrix.start[row];
        int rowEnd = matrix.start[row + 1];

        // compute running sum per thread
        values[threadIdx.x] = 0.0f;

        for (int j = rowStart + lane; j < rowEnd; j += WARP_SIZE) { // 可能一次完不成,eg 51列,warp需做两遍
            values[threadIdx.x] += matrix.values[j] * x[matrix.indices[j]];
        }

#if CUDART_VERSION > 9000
        __syncwarp();
#else
        __syncthreads();
#endif

        // parallel reduction in shared memory
        for (int d = WARP_SIZE >> 1; d >= 1; d >>= 1) {
            if (lane < d)
                values[threadIdx.x] += values[threadIdx.x + d];
#if CUDART_VERSION > 9000
            __syncwarp();
#else
            __syncthreads();
#endif
        }

        // first thread in a warp writes the result
        if (lane == 0) {
            y[row] = values[threadIdx.x];
        }
    }
}


void launch_csrMatVecMult(float *y, const DevCsrMatrix matrix, const float *x,
                       const ExecutionMode mode) {
    constexpr int TILE_SIZE = 64;
    switch (mode) {
        case ExecutionMode::PAGERANK: {
            // #threads = #rows (= N)
            // TODO: T3.2a define grid/block size
            // #threads = #rows (= N)
            dim3 grid(get1DGrid(TILE_SIZE, matrix.numRows), 1, 1);
            dim3 block(TILE_SIZE, 1, 1);
            kernel_csrMatVecMult<<<grid, block>>>(y, matrix, x);
            break;
        }
        case ExecutionMode::PAGERANK_VECTORIZED: {
            // TODO: H3.1 define grid/block size
            // #threads = #rows * #threads per row (= N * WARP_SIZE), each row done by a warp
            dim3 grid(get1DGrid(TILE_SIZE, matrix.numRows * WARP_SIZE), 1, 1);
            dim3 block(TILE_SIZE, 1, 1);
            kernel_csrMatVecMult_vectorized<TILE_SIZE><<<grid, block>>>(y, matrix, x);
            break;
        }
        default: {
            std::stringstream stream;
            stream << "Unknown execution mode #(" << mode << ") for page rank solver";
            throw std::runtime_error(stream.str());
        }
    }
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_ellMatVecMult(float *y, const DevEllMatrix matrix, const float *x) {
  // TODO: T4.1a
  // 一个thread负责一行的点积
    const int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < matrix.numRows) {
        float dot = 0.0f;
        for (int i = 0; i < matrix.numColsPerRow; ++i) {
            int column = matrix.indices[row + i * matrix.numRows];
            float value = matrix.values[row + i * matrix.numRows];
            if (value != 0.0f) {
                dot += value * x[column];
            }
        }
        y[row] = dot;
    }
}


void launch_ellMatVecMult(float *y, const DevEllMatrix matrix, const float *x) {
  // TODO: T4.1a
    constexpr int TILE_SIZE = 64;
    dim3 grid(get1DGrid(TILE_SIZE, matrix.numRows), 1, 1);
    dim3 block(TILE_SIZE, 1, 1);

    kernel_ellMatVecMult<<<grid, block>>>(y, matrix, x);
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_bandMatVecMult(float *y, const DevBandMatrix matrix, const float *x) {
  // TODO: H5.1
   const int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < matrix.numRows) {
        float dot = 0.0f;

        for (int k = 0; k < (2 * matrix.halfSize + 1); ++k) {
            float value = matrix.values[row + k * matrix.numRows];
            int column = row + k - matrix.halfSize;

            if ((column >= 0) && (column < matrix.numRows)) {
              if (value != 0) {
                  dot += value * x[column];
              }
            }
        }
        y[row] = dot;
    }
}


void launch_bandMatVecMult(float *y, const DevBandMatrix matrix, const float *x) {
    // TODO: H5.1
    constexpr int TILE_SIZE = 64;
    //#threads = #rows (= N)
    dim3 grid(get1DGrid(TILE_SIZE, matrix.numRows), 1, 1);
    dim3 block(TILE_SIZE, 1, 1);

    kernel_bandMatVecMult<<<grid, block>>>(y, matrix, x);
    CHECK_ERR;
}